#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "gemm_fp32.cuh"
#include "common/polybench.h"
#include "common/polybenchUtilFuncts.h"

#define GPU_DEVICE 0

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

/* Declared constant values for ALPHA and BETA */
#define ALPHA 32412.0f
#define BETA 2123.0f

#define RUN_ON_CPU

void gemm(int ni, int nj, int nk, fp32_t alpha, fp32_t beta, fp32_t POLYBENCH_2D(A,NI,NK,ni,nk), 
         fp32_t POLYBENCH_2D(B,NK,NJ,nk,nj), fp32_t POLYBENCH_2D(C,NI,NJ,ni,nj))
{
    int i, j, k;
    
    for (i = 0; i < _PB_NI; i++)
    {
        for (j = 0; j < _PB_NJ; j++)
        {
            C[i][j] *= beta;
            for (k = 0; k < _PB_NK; ++k)
            {
                C[i][j] += alpha * A[i][k] * B[k][j];
            }
        }
    }
}

void init(int ni, int nj, int nk, fp32_t* alpha, fp32_t* beta, fp32_t POLYBENCH_2D(A,NI,NK,ni,nk), 
        fp32_t POLYBENCH_2D(B,NK,NJ,nk,nj), fp32_t POLYBENCH_2D(C,NI,NJ,ni,nj))
{
    int i, j;

    *alpha = 32412.0f;
    *beta = 2123.0f;

    for (i = 0; i < ni; i++)
    {
        for (j = 0; j < nk; j++)
        {
            A[i][j] = ((fp32_t) i*j) / NI;
        }
    }

    for (i = 0; i < nk; i++)
    {
        for (j = 0; j < nj; j++)
        {
            B[i][j] = ((fp32_t) i*j) / NI;
        }
    }

    for (i = 0; i < ni; i++)
    {
        for (j = 0; j < nj; j++)
        {
            C[i][j] = ((fp32_t) i*j) / NI;
        }
    }
}

void compareResults(int ni, int nj, fp32_t POLYBENCH_2D(C,NI,NJ,ni,nj), 
                   fp32_t POLYBENCH_2D(C_outputFromGpu,NI,NJ,ni,nj))
{
    int i, j, fail;
    fail = 0;
    
    for (i=0; i < ni; i++) 
    {
        for (j=0; j < nj; j++) 
        {
            if (percentDiff(C[i][j], C_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
            {
                fail++;
            }
        }
    }
    printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", 
           PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

void GPU_argv_init()
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
    printf("Setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
    hipSetDevice(GPU_DEVICE);
}

__global__ void gemm_kernel_fp32(int ni, int nj, int nk, fp32_t alpha, fp32_t beta, 
                                fp32_t *a, fp32_t *b, fp32_t *c)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < ni && j < nj)
    {
        fp32_t temp = 0.0f;
        for (int k = 0; k < nk; k++)
        {
            temp += alpha * a[i * nk + k] * b[k * nj + j];
        }
        c[i * nj + j] = beta * c[i * nj + j] + temp;
    }
}

void gemmCuda_fp32(int ni, int nj, int nk, fp32_t alpha, fp32_t beta, 
                   fp32_t POLYBENCH_2D(A,NI,NK,ni,nk), 
                   fp32_t POLYBENCH_2D(B,NK,NJ,nk,nj), 
                   fp32_t POLYBENCH_2D(C,NI,NJ,ni,nj), 
                   fp32_t POLYBENCH_2D(C_outputFromGpu,NI,NJ,ni,nj))
{
    fp32_t *A_gpu;
    fp32_t *B_gpu;
    fp32_t *C_gpu;

    hipMalloc((void **)&A_gpu, sizeof(fp32_t) * NI * NK);
    hipMalloc((void **)&B_gpu, sizeof(fp32_t) * NK * NJ);
    hipMalloc((void **)&C_gpu, sizeof(fp32_t) * NI * NJ);
    
    hipMemcpy(A_gpu, A, sizeof(fp32_t) * NI * NK, hipMemcpyHostToDevice);
    hipMemcpy(B_gpu, B, sizeof(fp32_t) * NK * NJ, hipMemcpyHostToDevice);
    hipMemcpy(C_gpu, C, sizeof(fp32_t) * NI * NJ, hipMemcpyHostToDevice);
    
    dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
    dim3 grid((NJ + DIM_THREAD_BLOCK_X - 1) / DIM_THREAD_BLOCK_X,
              (NI + DIM_THREAD_BLOCK_Y - 1) / DIM_THREAD_BLOCK_Y);

    /* Start timer. */
    polybench_start_instruments;

    // Launch FP32-optimized kernel
    gemm_kernel_fp32<<< grid, block >>>(ni, nj, nk, alpha, beta, A_gpu, B_gpu, C_gpu);
    hipDeviceSynchronize();

    /* Stop and print timer. */
    printf("GPU Time in seconds (FP32):\n");
    polybench_stop_instruments;
    polybench_print_instruments;

    hipMemcpy(C_outputFromGpu, C_gpu, sizeof(fp32_t) * NI * NJ, hipMemcpyDeviceToHost);    
    
    hipFree(A_gpu);
    hipFree(B_gpu);
    hipFree(C_gpu);
}

int main(int argc, char *argv[])
{
    /* Retrieve problem size. */
    int ni = NI;
    int nj = NJ;
    int nk = NK;

    /* Variable declaration/allocation. */
    fp32_t alpha;
    fp32_t beta;
    POLYBENCH_2D_ARRAY_DECL(A,fp32_t,NI,NK,ni,nk);
    POLYBENCH_2D_ARRAY_DECL(B,fp32_t,NK,NJ,nk,nj);
    POLYBENCH_2D_ARRAY_DECL(C,fp32_t,NI,NJ,ni,nj);
    POLYBENCH_2D_ARRAY_DECL(C_outputFromGpu,fp32_t,NI,NJ,ni,nj);

    init(ni, nj, nk, &alpha, &beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C));
    
    GPU_argv_init();
    
    gemmCuda_fp32(ni, nj, nk, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), 
                  POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));

    #ifdef RUN_ON_CPU
        /* Start timer. */
        polybench_start_instruments;

        gemm(ni, nj, nk, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C));
        
        /* Stop and print timer. */
        printf("CPU Time in seconds:\n");
        polybench_stop_instruments;
        polybench_print_instruments;
    
        compareResults(ni, nj, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));
    #endif

    POLYBENCH_FREE_ARRAY(A);
    POLYBENCH_FREE_ARRAY(B);  
    POLYBENCH_FREE_ARRAY(C);  
    POLYBENCH_FREE_ARRAY(C_outputFromGpu); 

    return 0;
}

#include "common/polybench.c"