#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "drivers/dgemm_2d_tiled_driver.h" 
#include "kernels/dgemm_2d_tiled.cuh"

#define CUDA_CHECK(call)                                                          \
    ({                                                                            \
        hipError_t err = call;                                                   \
        if (err != hipSuccess) {                                                 \
            std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__          \
                      << " - " << hipGetErrorString(err) << " (" #call ")" << std::endl; \
        }                                                                         \
        err == hipSuccess; /* This is the value the macro expression returns */  \
    })

/// @brief Driver for 2D-Tiled DGEMM Kernel
/// @param M Number of rows in A
/// @param N Number of cols in B
/// @param K Number of cols in A and number of rows in B
/// @param hA Pointer to A matrix in host memory (M x K)
/// @param hB Pointer to B matrix in host memory (K x N)
/// @param hC Pointer to C matrix in host memory (M x N)
bool dgemm_2d_tiled_driver(int M, int N, int K, double* hA, double* hB, double* hC) {
  const unsigned int BM = 64;
  const unsigned int BK = 16;
  const unsigned int BN = 64;
  const unsigned int TM = 4;
  const unsigned int TN = 4;

  dim3 gridDim(N/BN, M/BM, 1);
  dim3 blockDim(BN/TN, BM/TM, 1);
  const size_t sharedMemSize = BK * (BM + BN) * sizeof(double);

  double *dA = nullptr, *dB = nullptr, *dC = nullptr;
  if(!CUDA_CHECK(hipMalloc(&dA, M * K * sizeof(double)))) goto cleanup;
  if(!CUDA_CHECK(hipMalloc(&dB, K * N * sizeof(double)))) goto cleanup;
  if(!CUDA_CHECK(hipMalloc(&dC, M * N * sizeof(double)))) goto cleanup;

  if(!CUDA_CHECK(hipMemcpy(dA, hA, M * K * sizeof(double), hipMemcpyHostToDevice))) goto cleanup;
  if(!CUDA_CHECK(hipMemcpy(dB, hB, K * N * sizeof(double), hipMemcpyHostToDevice))) goto cleanup;

  std::cout << "DRIVER: Launching 2D-Tiled Kernel..." << std::endl;
  dgemm_2d_tiled<BM, BK, BN, TM, TN><<<gridDim, blockDim, sharedMemSize>>>(M, N, K, dA, dB, dC);

  if (!CUDA_CHECK(hipGetLastError())) goto cleanup;
  if (!CUDA_CHECK(hipDeviceSynchronize())) goto cleanup;
  std::cout << "DRIVER: Kernel finished successfully." << std::endl;

  if(!CUDA_CHECK(hipMemcpy(hC, dC, M * N * sizeof(double), hipMemcpyDeviceToHost))) goto cleanup;

  cleanup:
  if(dA) hipFree(dA);
  if(dB) hipFree(dB);
  if(dC) hipFree(dC);

  return hipGetLastError() == hipSuccess;
}